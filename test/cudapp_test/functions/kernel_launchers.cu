#include "hip/hip_runtime.h"
//
// Created by developer on 3/24/20.
//

#include "cudapp/utilities/ide_helpers.h"

#include <>

#include "gtest/gtest.h"

#include "cudapp/functions/kernel_launchers.h"
#include "cudapp/memory/allocators/managed_allocator.h"

#include "cudapp_test/testing_helpers.h"

template <typename A, typename B>
__global__ void CopyValueKernel(A value, B out, unsigned int n) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    out[n] = value;
  }
}

using test_type = float;
static constexpr unsigned int num = 32;

TEST(KernelLauncher, CopyValue) {
  std::vector<test_type, cudapp::ManagedAllocator<test_type>> out(num, test_type{0});
  double val = 3.14;

  dim3 grid;
  dim3 block;
  auto function = CopyValueKernel<test_type, test_type*>;
  cudapp::LaunchFunction(grid, block, function, val, out.data(), out.size());
}